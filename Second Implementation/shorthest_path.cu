#include "hip/hip_runtime.h"
#define N 512

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <iostream>
#include <iomanip>
#include "../Lib/init.h"
#include "../Lib/verify.h"
#include "../Lib/cpu.h"
#include "../Lib/time_analysis.h"
#include "../Lib/copy.h"
#include "../Lib/print_matrix.h"
#include "floyd.h"

int main(int argc, char *argv[])
{
    // clock_t start, end;
    bool print_matrix_option;

    if (argv[1] == nullptr || strcmp(argv[1], "yes") != 0)
    {
        print_matrix_option = false;
    }
    else
    {
        print_matrix_option = true;
    }

    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    std::cout << "Device " << 0 << ": " << device_prop.name << '\n';
    hipSetDevice(0);

    size_t bytes = N * N * sizeof(float);

    std::cout << "Matrix memory occupation " << bytes << '\n';

    float *h_a;//, *h_cpu_a, *h_cpu_b;
    float *d_a;
    float time_gpu_tiled;//, time_cpu;

    h_a = (float *)malloc(bytes);
    // h_cpu_a = (float *)malloc(bytes);
    // h_cpu_b = (float *)malloc(bytes);

    init_data_random(h_a, N * N);

    if (print_matrix_option)
    {
        std::cout << "Initial distance matrix:\n";
        print_matrix(h_a, N * N);
    }

    // memset(h_cpu_b, 0, bytes);

    // copy_matrix(h_a, h_cpu_a, N * N);

    hipMalloc(&d_a, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

    // bool is_matrix_a_with_data = true;
    // start = clock();
    // for (int i = 1; i < N * 2; i = i << 1)
    // {
    //     if (is_matrix_a_with_data)
    //     {
    //         cpu_mmatrix(h_cpu_b, h_cpu_a, h_cpu_a, N);
    //         is_matrix_a_with_data = false;
    //     }
    //     else
    //     {
    //         cpu_mmatrix(h_cpu_a, h_cpu_b, h_cpu_b, N);
    //         is_matrix_a_with_data = true;
    //     }
    // }
    // end = clock();
    // if (!is_matrix_a_with_data)
    // {
    //     copy_matrix(h_cpu_b, h_cpu_a, N * N);
    // }

    // time_cpu = (double)(end - start) / CLOCKS_PER_SEC;

    int n_threads = 32;
    int n_blocks = N / n_threads;

    dim3 threads(n_threads, n_threads);
    dim3 blocks(n_blocks, n_blocks);

    std::cout << "Blocks: " << n_blocks << '\n';
    std::cout << "Threads/block: " << n_threads << '\n';
    std::cout << "Threads(total) " << n_threads * n_blocks << '\n';

    time_start();
    for (int k = 0; k < N; k++)
    {
        floyd<<<blocks,threads>>>(d_a, k);
        hipDeviceSynchronize();
    }
    time_end();

    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);
    time_gpu_tiled = elapsed_time;

    if (print_matrix_option)
    {
        // std::cout << "====================================\n";
        // print_matrix(h_cpu_a, N * N);
        // std::cout << "====================================\n";
        std::cout << "Resulting distance matrix:\n";
        print_matrix(h_a, N * N);
    }

    // check_results(h_cpu_a, h_a, N * N);

    // std::cout << "Time cpu: " << std::fixed << time_cpu << '\n';
    std::cout << "Time gpu: " << std::fixed << time_gpu_tiled << " ms" << '\n';

    // free(h_cpu_a);
    // free(h_cpu_b);
    free(h_a);
    hipFree(d_a);

    return 0;
}