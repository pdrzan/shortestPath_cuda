#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <iostream>
#include <iomanip> 
#include "../Lib/init.h"
#include "../Lib/verify.h"
#include "../Lib/cpu.h"
#include "../Lib/time_analysis.h"
#include "matrix_tiled.h"

#define N 4

int main(int argc, char *argv[])
{

    int device_id, number_of_sms;
    clock_t start, end;
    int n = atoi(argv[1]);
    bool print_matrix_option;

    if(argv[2] == nullptr || strcmp(argv[2], "yes") != 0)
    {
        print_matrix_option = false;
    }
    else
    {
        print_matrix_option = true;
    }

    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    printf("Device %D: %s \n", 0, device_prop.name);
    hipSetDevice(0);

    size_t bytes = n * n * sizeof(float);

    std::cout << "Matrix memory occupation" << bytes << '\n';

    float *h_a, *h_cpu, *h_tiled;
    float *d_a, *d_b, *d_tiled;
    float time_cpu, time_gpu_naive, time_gpu_tiled;

    h_a = (float *)malloc(bytes);
    // h_cpu = (float *)malloc(bytes);
    h_tiled = (float *)malloc(bytes);

    init_data_random(h_a, n * n);

    // memset(h_cpu, 0, bytes);
    memset(h_tiled, 0, bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_tiled, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_tiled, h_tiled, bytes, hipMemcpyHostToDevice);

    start = clock();
    cpu_mmatrix(h_cpu, h_a, h_a, n);
    end = clock();

    int n_threads = 2;
    int n_blocks = n / n_threads;

    dim3 threads(n_threads, n_threads);
    dim3 blocks(n_blocks, n_blocks);

    std::cout << "Blocks: " << n_blocks << '\n';
    std::cout << "Threads/block: " << n_threads << '\n';
    std::cout << "Threads(total) " << n_threads * n_blocks << '\n';

    time_start();
    matrixMul_tiled<<<blocks, threads>>>(d_a, n);
    hipDeviceSynchronize();
    time_end();

    hipMemcpy(h_tiled, d_tiled, bytes, hipMemcpyDeviceToHost);
    std::cout << "Time: " << elapsed_time << '\n';

    check_results(h_cpu, h_a, n * n);

    free(h_cpu); free(h_tiled); free(h_a);  
    hipFree(d_tiled); hipFree(d_a); hipFree(d_b);

    return 0;
}