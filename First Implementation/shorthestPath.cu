#include "hip/hip_runtime.h"
#define N 1024 // Matrix NxN

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "Lib/init.h"
#include "Lib/verify.h"
#include "Lib/cpu.h"
#include "matrix_tiled.h"

__global__ void ret_naive_matrixMultiply(float *A, float *B, float *C,
                                         int numARows, int numAColumns,
                                         int numBRows, int numBColumns,
                                         int numCRows, int numCColumns)
{
    //@@ Insert code to implement matrix multiplication here
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    if (numAColumns != numBRows)
        return;
    if ((Row < numARows) && (Col < numBColumns))
    {
        float Cvalue = 0;
        for (int k = 0; k < numAColumns; ++k)
            Cvalue += A[Row * numAColumns + k] * B[k * numBColumns + Col];
        C[Row * numCColumns + Col] = Cvalue;
    }
}

int main(int argc, char *argv[])
{

    int deviceId, numberOfSMs;
    clock_t start, end;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    printf("device %d: %s \n", 0, deviceProp.name);
    hipSetDevice(0);

    size_t bytes = N * N * sizeof(float);

    printf("Ocupação tamanho da Matriz %d\n", bytes);

    float *h_a, *h_b, *h_cpu, *h_naive, *h_tiled;
    float *d_a, *d_b, *d_naive, *d_tiled;
    float time_cpu, time_gpu_naive, time_gpu_tiled;

    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_cpu = (float *)malloc(bytes);
    h_naive = (float *)malloc(bytes);
    h_tiled = (float *)malloc(bytes);

    initDataRandom(h_a, N * N);
    initDataRandom(h_b, N * N);
    return 0;
}