#define N 1024 // Matrix NxN

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "../Lib/init.h"
#include "../Lib/verify.h"
#include "../Lib/cpu.h"
#include "matrix_tiled.h"

int main(int argc, char *argv[])
{

    int device_id, number_of_sms;
    clock_t start, end;

    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    printf("device %d: %s \n", 0, device_prop.name);
    hipSetDevice(0);

    size_t bytes = N * N * sizeof(float);

    printf("Matrix memory occupation %d\n", bytes);

    float *h_a, *h_b, *h_cpu, *h_naive, *h_tiled;
    float *d_a, *d_b, *d_naive, *d_tiled;
    float time_cpu, time_gpu_naive, time_gpu_tiled;

    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_cpu = (float *)malloc(bytes);
    h_tiled = (float *)malloc(bytes);

    init_data_random(h_a, N * N);
    init_data_random(h_b, N * N);

    return 0;
}