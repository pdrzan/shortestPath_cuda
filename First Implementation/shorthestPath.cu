#define N 1024 // Matrix NxN

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "../Lib/init.h"
#include "../Lib/verify.h"
#include "../Lib/cpu.h"
#include "matrix_tiled.h"

int main(int argc, char *argv[])
{

    int deviceId, numberOfSMs;
    clock_t start, end;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    printf("device %d: %s \n", 0, deviceProp.name);
    hipSetDevice(0);

    size_t bytes = N * N * sizeof(float);

    printf("Matrix memory occupation %d\n", bytes);

    float *h_a, *h_b, *h_cpu, *h_naive, *h_tiled;
    float *d_a, *d_b, *d_naive, *d_tiled;
    float time_cpu, time_gpu_naive, time_gpu_tiled;

    h_a = (float *)malloc(bytes);
    h_b = (float *)malloc(bytes);
    h_cpu = (float *)malloc(bytes);
    h_tiled = (float *)malloc(bytes);

    initDataRandom(h_a, N * N);
    initDataRandom(h_b, N * N);

    return 0;
}